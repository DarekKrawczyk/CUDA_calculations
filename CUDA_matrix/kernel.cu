﻿#include "hip/hip_runtime.h"

#include <iostream>
#include <string>

/*
* Kernel code, adding two matrices together A<-A+B
*/
__global__ void add(float* A, float* B, int M, int N) {

	int i = blockDim.x*blockIdx.x+threadIdx.x;
	int j = blockDim.y*blockIdx.y+threadIdx.y;
	B[i * M + j] += A[i * M + j];
}

void fill(float* tab, float value, int N, int M) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			tab[i * N + j] = value;
		}
	}
}

void show(float* tab, std::string text) {
	std::cout << text << '\n';
	for (int i = 0; i < sizeof(tab); i++) {
		for (int j = 0; j < sizeof(tab); j++) {
			std::cout << tab[i * sizeof(tab) + j] << ' ';
		}
		std::cout << '\n';
	}
	std::cout << '\n';
}

int main(){
	const int N = 10;
	const int M = 10;
	float* tab = new float[N*M];
	float* tab_ = new float[N*M];
	size_t size = N * M * sizeof(float);
	float* A_GPU;
	float* B_GPU;

	fill(tab, 5.0f, N, M);
	fill(tab_, 2.0f, N, M);

	show(tab,"A-Przed");
	show(tab_,"B-Przed");


	hipMalloc((void**)&A_GPU, size);
	hipMalloc((void**)&B_GPU, size);

	hipMemcpy(A_GPU, tab, size, hipMemcpyHostToDevice);
	hipMemcpy(B_GPU, tab_, size, hipMemcpyHostToDevice);

	dim3 block(5,5);
	dim3 threats(2,2);
	dim3 shared(threats.x, threats.y);

	add<<<block,threats >>>(A_GPU,B_GPU,M,N);

	hipMemcpy(tab, B_GPU, size, hipMemcpyDeviceToHost);

	show(tab,"Po");

	hipFree(A_GPU);
	hipFree(B_GPU);
	delete[]tab;
}
